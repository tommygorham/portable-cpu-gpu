
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloWorldKernel() {
    printf("Hello from GPU world!\n");
}

int main() {
    helloWorldKernel<<<1, 1>>>();
    hipDeviceSynchronize(); // Wait for the kernel to finish
    std::cout << "Hello from CPU world!" << std::endl;
    return 0;
}
