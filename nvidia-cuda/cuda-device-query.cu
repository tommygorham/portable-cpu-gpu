#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h> // Essential for CUDA Runtime API functions and structs

// A simple error checking macro for CUDA API calls
#define CUDA_CHECK(call)                                                         \
do {                                                                             \
    hipError_t err = call;                                                      \
    if (err != hipSuccess) {                                                    \
        std::cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__            \
                  << " - " << hipGetErrorString(err) << std::endl;            \
        exit(EXIT_FAILURE);                                                      \
    }                                                                            \
} while (0)

int main() {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return 1;
    }

    std::cout << "Found " << deviceCount << " CUDA device(s)." << std::endl;

    // Iterate through all found devices
    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t props;
        CUDA_CHECK(hipGetDeviceProperties(&props, dev));

        std::cout << "\n--- Device " << dev << ": " << props.name << " ---" << std::endl;

        // Mimicking HIP's architecture/compute capability output
        // For NVIDIA, this is typically major.minor for compute capability
        std::cout << "  Compute Capability (arch): " << props.major << "." << props.minor << std::endl;
        std::cout << "  Streaming Multiprocessors (SMs): " << props.multiProcessorCount << std::endl;
        std::cout << "  Global Memory:             " << props.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Shared Memory per Block:   " << props.sharedMemPerBlock / 1024 << " KB" << std::endl;
        std::cout << "  Registers per Block:       " << props.regsPerBlock << std::endl;
        std::cout << "  Warp Size:                 " << props.warpSize << " threads" << std::endl;
        std::cout << "  Max Threads per Block:     " << props.maxThreadsPerBlock << std::endl;
        std::cout << "  Max Block Dimensions:      (" << props.maxThreadsDim[0] << ", "
                  << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << ")" << std::endl;
        std::cout << "  Max Grid Dimensions:       (" << props.maxGridSize[0] << ", "
                  << props.maxGridSize[1] << ", " << props.maxGridSize[2] << ")" << std::endl;
        std::cout << "  Clock Rate (SM Clock):     " << props.clockRate / 1000 << " MHz" << std::endl;
        // CUDA's memoryClockRate is generally not directly exposed like HIP's,
        // but we can approximate memory bandwidth.
        // Assuming DDR, so effective clock rate is 2 * actual, then * bus width / 8 bits per byte
        // For GDDR5/6, effective is 4x for GDDR5 and 2x for GDDR6 vs base clock.
        // Props.memoryClockRate is often given as actual clock, not effective data rate for GDDR.
        // A more accurate bandwidth calculation for modern GPUs is complex and dependent on memory type.
        // We'll use a simpler approximation based on props.memoryBusWidth and props.clockRate for example.
        // However, props.memoryClockRate is not typically available directly in this struct.
        // We will output bus width which is a good indicator.
        std::cout << "  Memory Bus Width:          " << props.memoryBusWidth << " bits" << std::endl;

        // For Max memory allocation, `maxTexture1D` is a common proxy in CUDA too.
        // CUDA also has `l2CacheSize`, etc.
        // hipMalloc often aligns to specific sizes and usually gives you large contiguous blocks.
        // Let's use maxTexture1D as requested for consistency if it's meaningful for your card.
        // A more reliable way to get max allocation is `hipMemGetInfo` after allocating.
        std::cout << "  Max 1D Texture Size:       " << props.maxTexture1D / (1024 * 1024) << " MB (as proxy for max alloc)" << std::endl;

        std::cout << "  ECC Enabled:               " << (props.ECCEnabled ? "Yes" : "No") << std::endl;
        std::cout << "  Integrated GPU:            " << (props.integrated ? "Yes" : "No") << std::endl;
        std::cout << "  Async Engines:             " << props.asyncEngineCount << std::endl; // Specific to NVIDIA for overlap
    }

    return 0;
}
